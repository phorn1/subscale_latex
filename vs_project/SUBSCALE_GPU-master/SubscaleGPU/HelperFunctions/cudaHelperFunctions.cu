#include "cudaHelperFunctions.cuh"

// copies array form device to local host memory
template<typename arrayType>
void copyArrayDeviceToLocal(arrayType* target_arr, arrayType* source_arr, uint64_t arrSize)
{
	hipError_t cudaStatus = hipMemcpy(target_arr, source_arr, arrSize * sizeof(arrayType), hipMemcpyDeviceToHost);
	checkStatus(cudaStatus);
}

// copies array form local host to device memory
template<typename arrayType>
void copyArrayLocalToDevice(arrayType* target_arr, arrayType* source_arr, uint64_t arrSize)
{
	hipError_t cudaStatus = hipMemcpy(target_arr, source_arr, arrSize * sizeof(arrayType), hipMemcpyHostToDevice);
	checkStatus(cudaStatus);
}

// copies array form device to device memory
template<typename arrayType>
void copyArrayDeviceToDevice(arrayType* target_arr, arrayType* source_arr, uint64_t arrSize)
{
	hipError_t cudaStatus = hipMemcpy(target_arr, source_arr, arrSize * sizeof(arrayType), hipMemcpyDeviceToDevice);
	checkStatus(cudaStatus);
}

// synchronizes all running streams
void synchronizeKernelCall()
{
	hipError_t cudaStatus = hipDeviceSynchronize();
	checkStatus(cudaStatus);
}

// checks for an error in the execution of a CUDA method 
void checkStatus(hipError_t cudaStatus)
{
	if (cudaStatus != hipSuccess)
	{
		throw std::runtime_error(hipGetErrorString(cudaStatus));
	}
}


template void copyArrayDeviceToLocal<uint32_t>(uint32_t* target_arr, uint32_t* source_arr, uint64_t arrSize);
template void copyArrayDeviceToLocal<uint64_t>(uint64_t* target_arr, uint64_t* source_arr, uint64_t arrSize);
template void copyArrayLocalToDevice<uint32_t>(uint32_t* target_arr, uint32_t* source_arr, uint64_t arrSize);
template void copyArrayLocalToDevice<uint64_t>(uint64_t* target_arr, uint64_t* source_arr, uint64_t arrSize);
template void copyArrayDeviceToDevice<uint32_t>(uint32_t* target_arr, uint32_t* source_arr, uint64_t arrSize);
template void copyArrayDeviceToDevice<uint64_t>(uint64_t* target_arr, uint64_t* source_arr, uint64_t arrSize);